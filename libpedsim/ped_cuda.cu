#include "hip/hip_runtime.h"
//
// pedsim - A microscopic pedestrian simulation system.
// Copyright (c) 2003 - 2014 by Christian Gloor
//
//
// Adapted for Low Level Parallel Programming 2017
//
#include "ped_cuda.h"
#include "ped_waypoint.h"
#include <math.h>

#include <stdlib.h>
#include <iostream>

Ped::Cuagents::Cuagents(Ped::Tagents* agents_array) {
	//static float *restrict mat_a __attribute__((aligned (XMM_ALIGNMENT_BYTES)));
	this->size = agents_array->agents.size();

	//hipError_t cudaStatus;


    hipMallocManaged((void**)&this->x, sizeof(float) * agents_array->agents.size());
    hipMallocManaged((void**)&this->y, sizeof(float) * agents_array->agents.size());

    hipMallocManaged((void**)&this->dest_x, sizeof(float) * agents_array->agents.size());
    hipMallocManaged((void**)&this->dest_y, sizeof(float) * agents_array->agents.size());
    hipMallocManaged((void**)&this->dest_r, sizeof(float) * agents_array->agents.size());

    hipMallocManaged((void**)&this->waypoint_x, sizeof(float*) * agents_array->agents.size());
    hipMallocManaged((void**)&this->waypoint_y, sizeof(float*) * agents_array->agents.size());
    hipMallocManaged((void**)&this->waypoint_r, sizeof(float*) * agents_array->agents.size());
    hipMallocManaged((void**)&this->waypoint_ptr, sizeof(float) * agents_array->agents.size());
    hipMallocManaged((void**)&this->waypoint_len, sizeof(float) * agents_array->agents.size());

	hipMemcpy((void**)&this->x, (void**)&agents_array->x, sizeof(float) * agents_array->agents.size(), hipMemcpyHostToDevice);
	hipMemcpy((void**)&this->y, (void**)&agents_array->y, sizeof(float) * agents_array->agents.size(), hipMemcpyHostToDevice);

	hipMemcpy((void**)&this->waypoint_ptr, (void**)&agents_array->waypoint_ptr, sizeof(float) * agents_array->agents.size(), hipMemcpyHostToDevice);
	hipMemcpy((void**)&this->waypoint_len, (void**)&agents_array->waypoint_len, sizeof(float) * agents_array->agents.size(), hipMemcpyHostToDevice);



	for (int i = 0; i < agents_array->agents.size(); i++) {

        hipMallocManaged((void**)&this->waypoint_x[i], sizeof(float) * agents_array->waypoints[i]->size());
        hipMallocManaged((void**)&this->waypoint_y[i], sizeof(float) * agents_array->waypoints[i]->size());
        hipMallocManaged((void**)&this->waypoint_r[i], sizeof(float) * agents_array->waypoints[i]->size());

		hipMemcpy((void**)&this->waypoint_x[i], (void**)&agents_array->waypoint_x[i], sizeof(float) * agents_array->waypoints[i]->size(), hipMemcpyHostToDevice);
		hipMemcpy((void**)&this->waypoint_y[i], (void**)&agents_array->waypoint_y[i], sizeof(float) * agents_array->waypoints[i]->size(), hipMemcpyHostToDevice);
		hipMemcpy((void**)&this->waypoint_r[i], (void**)&agents_array->waypoint_r[i], sizeof(float) * agents_array->waypoints[i]->size(), hipMemcpyHostToDevice);
    }
}


void Ped::Cuagents::computeNextDesiredPosition(int i) {
	
	double diffX = dest_x[i] - this->x[i];
	double diffY = dest_y[i] - this->y[i];

	double len = sqrt(diffX * diffX + diffY * diffY);
	
	this->x[i] = (int)round(this->x[i] + diffX / len);
	this->y[i] = (int)round(this->y[i] + diffY / len);


	// If the destination is null, or if the agent has reached its destination, then we compute its new destination coordinates.
	if (len < this->dest_r[i]) {
		this->dest_x[i] = this->waypoint_x[i][this->waypoint_ptr[i]];
		this->dest_y[i] = this->waypoint_y[i][this->waypoint_ptr[i]];
		this->dest_r[i] = this->waypoint_r[i][this->waypoint_ptr[i]];

		this->waypoint_ptr[i] += 1;
		if (this->waypoint_ptr[i] == this->waypoint_len[i])
			this->waypoint_ptr[i] = 0;
	}



}

void Ped::Cuagents::free() {
	hipFree((void**)&this->x);
	hipFree((void**)&this->y);

	hipFree((void**)&this->dest_x);
	hipFree((void**)&this->dest_y);
	hipFree((void**)&this->dest_r);
	
	for (int i = 0; i < this->size; i++) {

		hipFree((void**)&this->waypoint_x[i]);
		hipFree((void**)&this->waypoint_y[i]);
		hipFree((void**)&this->waypoint_r[i]);

    }

	hipFree((void**)&this->waypoint_x);
	hipFree((void**)&this->waypoint_y);
	hipFree((void**)&this->waypoint_r);
	hipFree((void**)&this->waypoint_ptr);
	hipFree((void**)&this->waypoint_len);


}

