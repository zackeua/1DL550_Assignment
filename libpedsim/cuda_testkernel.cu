#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "ped_agents.h"
#include "ped_waypoint.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


__global__ void print_func() {
	printf("Hello world from thread %d\n", 
	blockIdx.x * blockDim.x + threadIdx.x);
}

void hello()
{
	print_func <<<2, 10>>> ();
}

/*
__global__ void cuda_func(int n, Ped::Tagents* agents_array) {
	int index = threadIdx.x;
	int stride = blockDim.x;

	for (int i = index; i < n; i += stride) {
		//agents_array->computeNextDesiredPosition(i);

		Ped::Twaypoint* nextDestination = NULL;
		bool agentReachedDestination = false;

		if (agents_array->destination[i] != NULL) {
			// compute if agent reached its current destination
			

			double diffX = agents_array->dest_x[i] - agents_array->x[i];
			double diffY = agents_array->dest_y[i] - agents_array->y[i];
			double length = sqrt(diffX * diffX + diffY * diffY);
			agentReachedDestination = length < agents_array->dest_r[i];
			//std::cout << " " << this->x[i] << std::endl;
			//std::cout << " " << this->y[i] << std::endl;
			//std::cout << length << " " << this->destination[i]->getr() << std::endl;
		}

		if ((agentReachedDestination || agents_array->destination[i] == NULL) && !agents_array->waypoints[i]->empty()) {
			// Case 1: agent has reached destination (or has no current destination);
			// get next destination if available
			if (agents_array->destination[i] != NULL) {
				agents_array->waypoints[i]->push_back(agents_array->destination[i]);
			}
			nextDestination = agents_array->waypoints[i]->front();
			agents_array->dest_x[i] = nextDestination->x;
			agents_array->dest_y[i] = nextDestination->y;
			agents_array->dest_r[i] = nextDestination->r;

			agents_array->waypoints[i]->pop_front();
			// DO NOT print destination here, might be NULL
		}
		else {
			// Case 2: agent has not yet reached destination, continue to move towards
			// current destination
			nextDestination = agents_array->destination[i];
		}

		agents_array->destination[i] = nextDestination;


		if (agents_array->destination[i] == NULL) {
			// no destination, no need to
			// compute where to move to
			return;
		}
		// Safe to print here
		//std::cout << this->destination[i]->getx() << std::endl;	
		//std::cout << this->destination[i]->gety() << std::endl;

		//double diffX = destination[i]->getx() - this->x[i];
		//double diffY = destination[i]->gety() - this->y[i];

		// SIMD: recleare diffX and diffY as simd
		double diffX = agents_array->dest_x[i] - agents_array->x[i];
		double diffY = agents_array->dest_y[i] - agents_array->y[i];

		double len = sqrt(diffX * diffX + diffY * diffY);
		
		// SIMD:
		agents_array->x[i] = (int)round(agents_array->x[i] + diffX / len);
		agents_array->y[i] = (int)round(agents_array->y[i] + diffY / len);

		//agents_array->agents[i]->setX(agents_array->x[i]);
		//agents_array->agents[i]->setY(agents_array->y[i]);

	}
}


int cuda_tick(Ped::Tagents* agents) {
	Ped::Tagents* cuda_agents;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMallocManaged((void**)&cuda_agents, sizeof(agents));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!\n");
		return 1;
	}

	int number_of_blocks = 1;
	int threads_per_block = 1;
	cuda_func <<<number_of_blocks, threads_per_block>>> (cuda_agents->agents.size(), cuda_agents);

	cudaStatus = hipDeviceSynchronize();

	hipFree(cuda_agents);

	return 0;

}
*/

int cuda_test()
{
    static int tested = 0;

	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

    if (tested == 1)
        return 0;
    tested = 1;

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!\n");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		fprintf(stderr, "%s.\n", hipGetErrorString(hipGetLastError()));
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>>(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else
	{
		//fprintf(stderr, "Cuda launch succeeded! \n");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	if (cudaStatus != 0){
		fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
	}
	else{
		fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing
	}

	return cudaStatus;
}
