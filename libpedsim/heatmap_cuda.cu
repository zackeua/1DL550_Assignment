#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//
#include "ped_model.h"
#include  "heatmap_cuda.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

// Allocationg memory on GPU
void Ped::Model::setupHeatmapCUDA() {
	hipMalloc((void**)&heatmap_cuda, sizeof(int) * SIZE * SIZE);
	hipMalloc((void**)&scaled_heatmap_cuda, sizeof(int) * SCALED_SIZE * SCALED_SIZE);
	hipMalloc((void**)&blurred_heatmap_cuda, sizeof(int) * SCALED_SIZE * SCALED_SIZE);
}

__global__ void fadeHeatmapCUDA(int* heatmap) {
	for (int i =  blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x)
		heatmap[i] = (int)round(heatmap[i] * 0.80);
}

__global__ void incrementHeatCUDA(int numberOfAgents, int* heatmap, float* desiredX, float* desiredY) {
	// Count how many agents want to go to each location
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numberOfAgents; i += blockDim.x * gridDim.x) {
		int x = desiredX[i];
		int y = desiredY[i];

		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
			continue;

		// intensify heat for better color results
		heatmap[y * SIZE + x] += 40;
		// printf("Heatmap[%f * SIZE + %f]: %d\n",desiredX[i], desiredY[i],  heatmap[y * SIZE + x]);
	}

}

__global__ void capHeatmapCUDA(int* heatmap) {
	for (int i =  blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x)
		heatmap[i] = heatmap[i] < 255 ? heatmap[i] : 255;
}

__global__ void scaledHeatmapCUDA(int* heatmap, int* scaled_heatmap) {
	// Scale the data for visual representation
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x) {
		int value = heatmap[i];
		for (int cellY = 0; cellY < CELLSIZE; cellY++)
			for (int cellX = 0; cellX < CELLSIZE; cellX++) {

				// scaled_heatmap[i + CELLSIZE * cellX + CELLSIZE * cellY] = value;
				scaled_heatmap[(i % SIZE) * CELLSIZE + cellX +
				                (i / SIZE) * SCALED_SIZE * CELLSIZE + cellY * SCALED_SIZE] = value;
			}
		// if (value != 0)
		// 	printf("scaled_heatmap[%d] = %d\n", i, value);
	}
}

__global__ void blurredHeatmapCUDA(int* scaled_heatmap, int* blurred_cuda) {
	//Weights for blur filter
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};

	#define WEIGHTSUM 273
	#define OFFSET SCALED_SIZE * 2 + 2
	// // Apply Gaussian blurfilter
	// for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (SCALED_SIZE -2) * (SCALED_SIZE-2); i += blockDim.x * gridDim.x) {
	// // for (int i = 0; i < (SCALED_SIZE - 2) * (SCALED_SIZE - 2); i++) {
	// 	int sum = 0;
	// 	for (int k = -2; k < 3; k++)
	// 		for (int l = -2; l < 3; l++)
	// 			sum += w[2 + k][2 + l] * scaled_heatmap[OFFSET + i % (SCALED_SIZE - 2) + l + k * SCALED_SIZE];


	// 	int value = sum / WEIGHTSUM;
	// 	blurred_cuda[i] = 0x00FF0000 | value << 24;
	// }
	// for (int block = blockIdx.x; block < 2; block += 1) {
	// 	int* shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));

	// 	__shared__ int** heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	// 	for (int i = 0; i < SCALED_SIZE; i++)
	// 	{
	// 		heatmap[i] = shm + SCALED_SIZE*i;
	// 	}

	int block = blockIdx.x;

	__shared__ int heatmap[SCALED_SIZE*SCALED_SIZE/blockDim.x + SCALED_SIZE*4 +4];
	for (int i = 0; i < SCALED_SIZE*SCALED_SIZE/blockDim.x + SCALED_SIZE*4 +4; i++) {
		int offset = -2 * SCALED_SIZE;
		if (block == 0) {
			offset = 0;
		} else if (block == blockDim.x -1) {
			offset = -4 * SCALED_SIZE;
		}
		heatmap[i] = scaled_heatmap[i +SCALED_SIZE*SCALED_SIZE/blockDim.x * block + offset];
	}



		// Apply Gaussian blurfilter
	for (int i = blockDim.x + threadIdx.x; i < (SCALED_SIZE) * (SCALED_SIZE); i += blockDim.x) {
		if (i < SCALED_SIZE * 2 || i > SCALED_SIZE * (SCALED_SIZE-2) || i % SCALED_SIZE < 2 || i % SCALED_SIZE > (SCALED_SIZE - 2))
			continue;
		
		int sum = 0;
		for (int k = -2; k < 3; k++)
			for (int l = -2; l < 3; l++) {
				int offset = 2 * SCALED_SIZE;
				if (block == 0) {
					offset = 0;
				} else if (block == blockDim.x -1) {
					offset = 4 * SCALED_SIZE;
				}
				sum += w[2 + k][2 + l] * heatmap[offset + i + l + k * SCALED_SIZE];
				//sum += w[2 + k][2 + l] * scaled_heatmap[i + l + k * SCALED_SIZE];
			}


		int value = sum / WEIGHTSUM;
		blurred_cuda[i] = 0x00FF0000 | value << 24;
	}
	// }

}

// Updates the heatmap according to the agent positions
void Ped::Model::updateHeatmapCUDA() {
	// Setting the number of threads
	int number_of_blocks = 10;
	int threads_per_block = 10;
	
	// Allocating the CUDA status
	hipError_t cudaStatus;

	// Setting the CUDA device
	cudaStatus = hipSetDevice(0);

	// Checking if that worked
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSetDevice0 failed!\n");
		return;
	}

	fadeHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->heatmap_cuda);

	// // Synchronizing the threads
	// cudaStatus = hipDeviceSynchronize();

	// // Checking if that worked
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "cudaSetDevice1 failed!\n");
	// 	fprintf(stderr, "%d\n", cudaStatus);
	// 	return;
	// }

	hipMemcpy(this->cuda_array.desiredX, this->agents_array->desiredX, this->agents.size() * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(this->cuda_array.desiredY, this->agents_array->desiredY, this->agents.size() * sizeof(float), hipMemcpyHostToDevice);

	incrementHeatCUDA <<<number_of_blocks, threads_per_block>>> (this->agents.size(), this->heatmap_cuda, this->cuda_array.desiredX, this->cuda_array.desiredY);

	// // Synchronizing the threads
	// cudaStatus = hipDeviceSynchronize();

	// // Checking if that worked
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "cudaSetDevice2 failed!\n");
	// 	fprintf(stderr, "%d\n", cudaStatus);
		
	// 	return;
	// }

	capHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->heatmap_cuda);
	
	// // Synchronizing the threads
	// cudaStatus = hipDeviceSynchronize();

	// // Checking if that worked
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "cudaSetDevice3 failed!\n");
	// 	fprintf(stderr, "%d\n", cudaStatus);
		
	// 	return;
	// }

	scaledHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->heatmap_cuda, this->scaled_heatmap_cuda);
	
	// // Synchronizing the threads
	// cudaStatus = hipDeviceSynchronize();
	
	// // Checking if that worked
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "cudaSetDevice4 failed!\n");
	// 	fprintf(stderr, "%d\n", cudaStatus);
		
	// 	return;
	// }

	blurredHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->scaled_heatmap_cuda, this->blurred_heatmap_cuda);

	// // Synchronizing the threads
	// cudaStatus = hipDeviceSynchronize();
	
	// // Checking if that worked
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "cudaSetDevice5 failed!\n");
	// 	fprintf(stderr, "%d\n", cudaStatus);
		
	// 	return;
	// }

	// hipMemcpy(this->blurred_heatmap[0], this->scaled_heatmap_cuda, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(this->blurred_heatmap[0], this->blurred_heatmap_cuda, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
	int i;
	std::cin >> i;

}
