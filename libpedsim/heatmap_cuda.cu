#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//
#include "ped_model.h"
#include  "heatmap_cuda.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

// Sets up the heatmap
void Ped::Model::setupHeatmapCUDA() {
	int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	for (int i = 0; i < SCALED_SIZE; i++) {
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}

	// Allocationg memory on GPU
	hipMalloc((void**)&heatmap, sizeof(int) * SIZE * SIZE);
	hipMalloc((void**)&scaled_heatmap, sizeof(int) * SCALED_SIZE * SCALED_SIZE);
	hipMalloc((void**)&blurred_cuda, sizeof(int) * SCALED_SIZE * SCALED_SIZE);
}

__global__ void fadeHeatmapCUDA(int* heatmap) {
	for (int i =  blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x)
		heatmap[i] = (int)round(heatmap[i] * 0.80);
}

__global__ void incrementHeatCUDA(int numberOfAgents, int* heatmap, float* desiredX, float* desiredY) {
	// Count how many agents want to go to each location
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numberOfAgents; i += blockDim.x * gridDim.x) {
		int x = desiredX[i];
		int y = desiredY[i];

		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
			continue;

		// intensify heat for better color results
		heatmap[y * SIZE + x] += 40;
	}
}

__global__ void capHeatmapCUDA(int* heatmap) {
	for (int i =  blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x)
		heatmap[i] = heatmap[i] < 255 ? heatmap[i] : 255;
}

__global__ void scaledHeatmapCUDA(int* heatmap, int* scaled_heatmap) {
	// Scale the data for visual representation
	for (int i =  blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x) {
		int value = heatmap[i];
		for (int cellY = 0; cellY < CELLSIZE; cellY++)
			for (int cellX = 0; cellX < CELLSIZE; cellX++)
				scaled_heatmap[(i % SIZE) * CELLSIZE + cellX +
				               (i / SIZE) * SCALED_SIZE + cellY * SCALED_SIZE] = value;
	}
}

__global__ void blurredHeatmapCUDA(int* blurred_cuda, int* scaled_heatmap) {
	// Weights for blur filter
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};

	#define WEIGHTSUM 273
	#define OFFSET SCALED_SIZE * 2 + 2
	// Apply Gaussian blurfilter
	for (int i = 0; i < (SCALED_SIZE - 2) * (SCALED_SIZE - 2); i++) {
		int sum = 0;
		for (int k = -2; k < 3; k++)
			for (int l = -2; l < 3; l++)
				sum += w[2 + k][2 + l] * scaled_heatmap[OFFSET + i % (SCALED_SIZE - 2) + l + k * SCALED_SIZE];

		int value = sum / WEIGHTSUM;
		blurred_cuda[i] = 0x00FF0000 | value << 24;
	}
}

// Updates the heatmap according to the agent positions
void Ped::Model::updateHeatmapCUDA() {
	// Setting the number of threads
	int number_of_blocks = 100;
	int threads_per_block = 100;
	
	fadeHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (*this->heatmap);

	incrementHeatCUDA <<<number_of_blocks, threads_per_block>>> (this->agents.size(), *this->heatmap, this->cuda_array.desiredX, this->cuda_array.desiredY);

	capHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (*this->heatmap);
	
	scaledHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (*this->heatmap, *this->scaled_heatmap);
	
	blurredHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (*this->blurred_cuda, *this->scaled_heatmap);

	hipMemcpy(this->blurred_heatmap, this->blurred_cuda, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
}
