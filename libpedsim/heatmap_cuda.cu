#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//
#include "ped_model.h"
#include  "heatmap_cuda.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

// Allocationg memory on GPU
void Ped::Model::setupHeatmapCUDA() {
	hipMalloc((void**)&heatmap_cuda, sizeof(int) * SIZE * SIZE);
	hipMalloc((void**)&scaled_heatmap_cuda, sizeof(int) * SCALED_SIZE * SCALED_SIZE);
	hipMalloc((void**)&blurred_heatmap_cuda, sizeof(int) * SCALED_SIZE * SCALED_SIZE);
}

__global__ void fadeHeatmapCUDA(int* heatmap) {
	for (int i =  blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x)
		heatmap[i] = (int)round(heatmap[i] * 0.80);
}

__global__ void incrementHeatCUDA(int numberOfAgents, int* heatmap, float* desiredX, float* desiredY) {
	// Count how many agents want to go to each location
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numberOfAgents; i += blockDim.x * gridDim.x) {
		int x = desiredX[i];
		int y = desiredY[i];

		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
			continue;

		// intensify heat for better color results
		heatmap[y * SIZE + x] += 40;
		// printf("Heatmap[%f * SIZE + %f]: %d\n",desiredX[i], desiredY[i],  heatmap[y * SIZE + x]);
	}

}

__global__ void capHeatmapCUDA(int* heatmap) {
	for (int i =  blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x)
		heatmap[i] = heatmap[i] < 255 ? heatmap[i] : 255;
}

__global__ void scaledHeatmapCUDA(int* heatmap, int* scaled_heatmap) {
	// Scale the data for visual representation
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < SIZE * SIZE; i += blockDim.x * gridDim.x) {
		int value = heatmap[i];
		for (int cellY = 0; cellY < CELLSIZE; cellY++)
			for (int cellX = 0; cellX < CELLSIZE; cellX++) {

				// scaled_heatmap[i + CELLSIZE * cellX + CELLSIZE * cellY] = value;
				scaled_heatmap[(i % SIZE) * CELLSIZE + cellX +
				                (i / SIZE) * SCALED_SIZE * CELLSIZE + cellY * SCALED_SIZE] = value;
			}
		// if (value != 0)
		// 	printf("scaled_heatmap[%d] = %d\n", i, value);
	}
}

__global__ void blurredHeatmapCUDA(int* scaled_heatmap, int* blurred_cuda) {
	//Weights for blur filter
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};

	#define WEIGHTSUM 273
	#define OFFSET SCALED_SIZE * 2 + 2
	// // Apply Gaussian blurfilter
	// for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (SCALED_SIZE -2) * (SCALED_SIZE-2); i += blockDim.x * gridDim.x) {
	// // for (int i = 0; i < (SCALED_SIZE - 2) * (SCALED_SIZE - 2); i++) {
	// 	int sum = 0;
	// 	for (int k = -2; k < 3; k++)
	// 		for (int l = -2; l < 3; l++)
	// 			sum += w[2 + k][2 + l] * scaled_heatmap[OFFSET + i % (SCALED_SIZE - 2) + l + k * SCALED_SIZE];


	// 	int value = sum / WEIGHTSUM;
	// 	blurred_cuda[i] = 0x00FF0000 | value << 24;
	// }

		// Apply Gaussian blurfilter
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (SCALED_SIZE) * (SCALED_SIZE); i += blockDim.x * gridDim.x) {
		if (i < SCALED_SIZE * 2 || i > SCALED_SIZE * (SCALED_SIZE-2) || i % SCALED_SIZE < 2 || i % SCALED_SIZE > (SCALED_SIZE - 2))
			continue;
		

		int sum = 0;
		for (int k = -2; k < 3; k++)
			for (int l = -2; l < 3; l++)
				sum += w[2 + k][2 + l] * scaled_heatmap[i + l + k * SCALED_SIZE];


		int value = sum / WEIGHTSUM;
		blurred_cuda[i] = 0x00FF0000 | value << 24;
	}


}

// Updates the heatmap according to the agent positions
void Ped::Model::updateHeatmapCUDA() {
	// Setting the number of threads
	int number_of_blocks = 10;
	int threads_per_block = 10;
	
	// Allocating the CUDA status
	hipError_t cudaStatus;

	// Setting the CUDA device
	cudaStatus = hipSetDevice(0);

	// Checking if that worked
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSetDevice0 failed!\n");
		return;
	}

	fadeHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->heatmap_cuda);

	// Synchronizing the threads
	cudaStatus = hipDeviceSynchronize();

	// Checking if that worked
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSetDevice1 failed!\n");
		fprintf(stderr, "%d\n", cudaStatus);
		return;
	}

	hipMemcpy(this->cuda_array.desiredX, this->agents_array->desiredX, this->agents.size() * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(this->cuda_array.desiredY, this->agents_array->desiredY, this->agents.size() * sizeof(float), hipMemcpyHostToDevice);

	incrementHeatCUDA <<<number_of_blocks, threads_per_block>>> (this->agents.size(), this->heatmap_cuda, this->cuda_array.desiredX, this->cuda_array.desiredY);

	// Synchronizing the threads
	cudaStatus = hipDeviceSynchronize();

	// Checking if that worked
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSetDevice2 failed!\n");
		fprintf(stderr, "%d\n", cudaStatus);
		
		return;
	}

	capHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->heatmap_cuda);
	
	// Synchronizing the threads
	cudaStatus = hipDeviceSynchronize();

	// Checking if that worked
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSetDevice3 failed!\n");
		fprintf(stderr, "%d\n", cudaStatus);
		
		return;
	}

	scaledHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->heatmap_cuda, this->scaled_heatmap_cuda);
	
	// Synchronizing the threads
	cudaStatus = hipDeviceSynchronize();
	
	// Checking if that worked
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSetDevice4 failed!\n");
		fprintf(stderr, "%d\n", cudaStatus);
		
		return;
	}

	blurredHeatmapCUDA <<<number_of_blocks, threads_per_block>>> (this->scaled_heatmap_cuda, this->blurred_heatmap_cuda);

	// Synchronizing the threads
	cudaStatus = hipDeviceSynchronize();
	
	// Checking if that worked
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSetDevice5 failed!\n");
		fprintf(stderr, "%d\n", cudaStatus);
		
		return;
	}

	// hipMemcpy(this->blurred_heatmap[0], this->scaled_heatmap_cuda, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(this->blurred_heatmap[0], this->blurred_heatmap_cuda, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
	int i;
	std::cin >> i;

}
